#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>

#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) { *pret = 42; }
// hipMemcpyDeviceToHost
// hipMemcpyHostToDevice
// hipMemcpyDeviceToDevice
int main() {
  int *pret;
  checkCudaErrors(hipMalloc(&pret, sizeof(int)));
  kernel<<<1, 1>>>(pret);
  checkCudaErrors(hipDeviceSynchronize());

  int ret;
  checkCudaErrors(hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost));
  printf("result: %d\n", ret);

  hipFree(pret);
  return 0;
}
