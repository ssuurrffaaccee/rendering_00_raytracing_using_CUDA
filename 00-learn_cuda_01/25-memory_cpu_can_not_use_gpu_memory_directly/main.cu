#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int *pret;
    checkCudaErrors(hipMalloc(&pret, sizeof(int)));
    kernel<<<1, 1>>>(pret);
    checkCudaErrors(hipDeviceSynchronize());
    printf("result: %d\n", *pret);// failed, cpu use gpu memory
    hipFree(pret);
    return 0;
}
