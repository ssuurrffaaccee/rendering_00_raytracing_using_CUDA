#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ int kernel() { // compile failed
    return 42;
}

int main() {
    int ret = kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    printf("%d\n", ret);
    return 0;
}
