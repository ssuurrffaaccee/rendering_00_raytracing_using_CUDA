#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>

#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) { *pret = 42; }

int main() {
  int *pret;
  checkCudaErrors(hipMallocManaged(&pret, sizeof(int)));
  kernel<<<1, 1>>>(pret);  // access by gpu
  checkCudaErrors(hipDeviceSynchronize());
  printf("result: %d\n",
         *pret);  // access by cpu, auto copied fron gpu by cuda context
  hipFree(pret);
  return 0;
}
