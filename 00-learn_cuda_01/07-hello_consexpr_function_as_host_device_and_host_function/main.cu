#include <cstdio>
#include <hip/hip_runtime.h>

// need --expt-relaxed-constexpr
constexpr const char *cuthead(const char *p) {
    return p + 1;
}

__global__ void kernel() {
    printf(cuthead("Gello, world!\n"));
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    printf(cuthead("Cello, world!\n"));
    return 0;
}
