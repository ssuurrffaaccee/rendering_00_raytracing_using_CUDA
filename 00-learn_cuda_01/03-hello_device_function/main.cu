#include <cstdio>
#include <hip/hip_runtime.h>

__device__ __inline__ void say_hello() {
    printf("Hello, world!\n");
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();//stop, then wait gpu to finish
    return 0;
}
