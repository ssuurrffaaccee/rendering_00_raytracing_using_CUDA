#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <vector>

#include "hip/hip_runtime_api.h"

template <class T>
struct CudaAllocator {
  using value_type = T;
  
  CudaAllocator() = default;
  template<class _Other>
  constexpr CudaAllocator(const CudaAllocator<_Other>&) noexcept {}
  
  T *allocate(size_t size) {
    T *ptr = nullptr;
    checkCudaErrors(hipMallocManaged(&ptr, size * sizeof(T)));
    return ptr;
  }

  void deallocate(T *ptr, size_t size = 0) { checkCudaErrors(hipFree(ptr)); }

  template <class... Args>
  void construct(T *p, Args &&...args) {
    if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>)) {
      // init when no arg or is_pod
      // unified memory so cpu can access
      // copy data from gpu to cpu, it's slow
      ::new ((void *)p) T(std::forward<Args>(args)...);
    }
  }
};

template <class Func>
__global__ void parallel_for(int n, Func func) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        func(i);
    }
}

int main() {
    int n = 65536;

    parallel_for<<<32, 128>>>(n, [] __device__ (int i) {
        printf("number %d\n", i);
    });

    checkCudaErrors(hipDeviceSynchronize());

    return 0;
}

